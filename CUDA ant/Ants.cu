#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <GL/glut.h>
#include "space.h"
#include <math.h>
#include <time.h>
using namespace std;

int gRand(int);
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__global__ void bug(int cur_row, int cur_col, int d_mrow, int d_mcol, space* d_board, bool path, int dir)
{
	int id =threadIdx.x;
	//printf("%d %d \n", id, id-d_mcol);
	//id is position
	//id +1 = next col over
	//id -1 is to the left
	// id+d_mcol is one row down

	double sN =0;
	double sE =0;
	double sS =0;
	double sW =0;

	bool N =false;
	bool S = false;
	bool E = false;
	bool W = false;


		if(id-d_mcol >= 0)
		{
			if(!d_board[id-d_mcol].is_blocked)
				sN = d_board[id-d_mcol].scent;
			else
				sN = -1;
		}
		if(id+1 < d_mcol)
		{
			if(!d_board[id+1].is_blocked)
				sE = d_board[(id+1)].scent;
			else
				sE = -1;
		}
		if(id+d_mcol < d_mrow)
		{
			if(!d_board[id+d_mcol].is_blocked)
				sS = d_board[id+d_mcol].scent;
			else
				sS = -1;
		}
		if(id-1 >=0)
		{
			if(!d_board[id-1].is_blocked)
				sW = d_board[id-1].scent;
			else
				sW = -1;
		}
		if(sN > sS)
			N = true;
		else
			S = true;
		if(sE > sW)
			E = true;
		else
			W = true;

		if(N && E)
		{
			if(sN > sE)
			{
				N = true;
				E = false;
			}
			else
			{
				N = false;
				E = true;
			}
		}
		else if(N && W)
		{
			if(sN > sW)
			{
				N = true;
				W = false;
			}
			else
			{
				N = false;
				W = true;
			}
		}
		else if (S && E)
		{
			if(sS > sE)
			{
				S = true;
				E = false;
			}
			else
			{
				E = true;
				S = false;
			}
		}
		else
		{
			if(sS > sW)
			{
				S = true;
				W = false;
			}
			else
			{
				S = false;
				W = true;
			}
		}
	if(d_board[id].num_ants > 0)
	{
		double chance = d_board[id].scent;
		double go = dir%((int)chance);
		if(dir%5 ==0)
		{
			if(dir%4 == 0 && sN > 0)
			{
				int *loc = &(d_board[id].num_ants);
				atomicSub(loc, 1);
				(d_board[id].scent)++;
				loc = &(d_board[id-d_mcol].num_ants);
				atomicAdd(loc, 1);
				//printf("north");
			}
			else if(dir%4 == 1 && sE > 0)
			{
				int *loc = &(d_board[id].num_ants);
				atomicSub(loc, 1);
				(d_board[id].scent)++;
				loc = &(d_board[id+1].num_ants);
				atomicAdd(loc, 1);
				//	printf("east");
			}
			else if(dir%4 == 2 && sS > 0)
			{
				int *loc = &(d_board[id].num_ants);
				atomicSub(loc, 1);

				loc = &(d_board[id+d_mcol].num_ants);
				atomicAdd(loc, 1);
				//printf("south");
			}
			else if(dir%4 == 3 && sW > 0)
			{
				int *loc = &(d_board[id].num_ants);
				atomicSub(loc, 1);
				(d_board[id].scent)++;

				loc = &(d_board[id-1].num_ants);
				atomicAdd(loc, 1);
				//	printf("west");
			}
			else
			{

			}
		}
		if(N)
		{
			int *loc = &(d_board[id].num_ants);

			atomicSub(loc, 1);
			(d_board[id].scent)++;
			loc = &(d_board[id-d_mcol].num_ants);
			atomicAdd(loc, 1);
			//printf("north");

		}
		else if(S)
		{
			int *loc = &(d_board[id].num_ants);
			atomicSub(loc, 1);
			(d_board[id].scent)++;

			loc = &(d_board[id+d_mcol].num_ants);
			atomicAdd(loc, 1);
			//printf("south");
		}

		else if(E)
		{
			int *loc = &(d_board[id].num_ants);
			atomicSub(loc, 1);
			(d_board[id].scent)++;
			loc = &(d_board[id+1].num_ants);
			atomicAdd(loc, 1);
		//	printf("east");
		}
		else if(W)
		{
			int *loc = &(d_board[id].num_ants);
			atomicSub(loc, 1);
			(d_board[id].scent)++;
			loc = &(d_board[id-1].num_ants);
			atomicAdd(loc, 1);
		//	printf("west");

		}
		else
		{
			printf("WAT /n");
		}

		printf("%d moved to %d %d\n", id, cur_row, cur_col);

		if((d_board[(cur_row)*(d_mcol)+cur_col].food_count >=0))
		{
			int *locfood = &(d_board[(cur_row)*(d_mcol)+cur_col].food_count);
			d_board[(cur_row)*(d_mcol)+cur_col].food_count=atomicSub(locfood, 1);

		}

	//printf("food left: %d\n", d_board[(cur_row)*(d_mcol)+cur_col].food_count);
	}

}


int main(int argc,char** argv) {

	printf("Hello Wrold\n");
	// these valuse dictate most of the behaviors.
	int row = 10;
	int col = 10;
	int start_ants = 4;	//this many ants thrown at the board to begin with

	srand(time(NULL));
	int hr = gRand(row/2); // row pos of home space make it random
	int hc = gRand(col/2); // col pos of home space make it random
	int br = gRand(row); // row pos of blocked start point. make it random
	int bc = gRand(col); // col pos of blocked start point. make it random


	int x = gRand(row-1);	//address the one d array with 2 d offset values.
	int y = gRand(col-1);

	//*** address formula cats[xpos*col+ypos]

	// board created below

	space* cats = new space[row*col];

	// populate the board
	cats[hr*col+hc].set_home();
	cats[(row-1)*col+(col-1)].food_mod(1000); // hypothetically, passing a negative number will decriment food count.
	cats[hr*col+hc].many_ants(start_ants);		//works


	if(br>= hr && bc >= hc)
	{
		int count = 3;
		for(int i = br; i < row; i++)
		{
				int roll = gRand(10);
				if(roll == 5 && count > 0)
				{
					count--;
				}
				else
				{
					cats[(i)*col+(bc)].waller();
				}

		}
		count = 3;
		for(int i = bc; i < col; i++)
		{
			int roll = gRand(10);
			printf("%d ", roll);
			if(roll == 5 && count > 0)
			{
				count--;
			}
			else
			{
				cats[(br)*col+(i)].waller();
			}
		}
		count = 3;

	}
		// bugs launched knowing home position, max rows, max cols, and the board in shared memory

	for(int i = 0; i < row; i++)
	{
		for(int j = 0; j < col; j++)
		{
			int pos = i*col+j;
			double s = sqrt((i*i)+(j*j));
			cats[pos].scent = s;
		}
	}
	for(int i = 0; i< row; i++)
			{
				for(int j = 0; j < col; j++)
				{
					printf("%d ",cats[i*col+j].num_ants);
				}
				printf("\n");
			}

	printf("%d \n", cats[(row-1)*col+(col-1)].food_count);
	printf("\nhome: %d,%d wallstart: %d, %d\n", hr, hc, br, bc);
	int count = 200;
//	while(cats[(row-1)*col+(col-1)].food_count ==1000)
	while(count > 0)
	{
	int* cur_row;
	int* cur_col;
	int* d_row;
	int* d_col;

	space* d_board;
	space* h_board = &cats[0];


	CUDA_CHECK_RETURN(hipMalloc(&cur_col, sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc(&d_row, sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc(&d_col, sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc(&cur_row, sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc(&d_board, sizeof(space)*row*col));


	CUDA_CHECK_RETURN(hipMemcpy(cur_row, &hr, sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(cur_col, &hc, sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_row, &row, sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_col, &col, sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_board, h_board, sizeof(space)*row*col, hipMemcpyHostToDevice));
	dim3 grid(1, 1, 1);
	dim3 block(10, 10, 1);
	bool path = true;
	int dir = gRand(101);
	bug<<<1, row*col>>>(hr, hc, row, col, d_board, path, dir);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launcint a = board[0].get_ants();hed work to complete
	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(hipMemcpy(h_board, d_board, sizeof(space)*row*col, hipMemcpyDeviceToHost));
	//CUDA_CHECK_RETURN(hipMemcpy(&posr, cur_row, sizeof(int), hipMemcpyDeviceToHost));
	//CUDA_CHECK_RETURN(hipMemcpy(&posc, cur_col, sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree((void*)cur_row));
	CUDA_CHECK_RETURN(hipFree((void*)cur_col));
	CUDA_CHECK_RETURN(hipFree((void*)d_row));
	CUDA_CHECK_RETURN(hipFree((void*)d_col));
	CUDA_CHECK_RETURN(hipDeviceReset());
	count--;
	for(int i = 0; i< row; i++)
			{
				for(int j = 0; j < col; j++)
				{
					if(cats[i*col+j].num_ants < 0)
					{
						cats[i*col+j].num_ants = 0;
					}
					if(cats[i*col+j].num_ants < 1)
					{
						cats[i*col+j].scent = (cats[i*col+j].scent)*.90;
					}
				}
			}
	}

	for(int i = 0; i< row; i++)
		{
			for(int j = 0; j < col; j++)
			{
				if(cats[i*col+j].is_blocked)
				{
					printf("B ");
				}
				else
				printf("%d ",(int)cats[i*col+j].scent);
			}
			printf("\n");
		}
	//printf("\n %d %d\n", posr, posc);
	return 0;
}

int gRand(int max){

	int r = (int)rand()%max;
	return r;
}


